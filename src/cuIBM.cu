/***************************************************************************//**
 * \mainpage cuIBM
 *
 *		A GPU-based Immersed Boundary Method
 *
 * \author Anush Krishnan (anush@bu.edu)
 */


/***************************************************************************//**
 * \file cuIBM.cu
 * \author Anush Krishnan (anush@bu.edu)
 * \author Christopher Minar (minarc@oregonstate.edu)
 * \brief Main source-file of \c cuIBM.
 */

#include "domain.h"
#include "io/io.h"
#include "solvers/NavierStokes/NavierStokesSolver.h"
#include "solvers/NavierStokes/luo_iter.h"
#include "solvers/NavierStokes/fadlunModified.h"
#include "solvers/NavierStokes/luoIBM.h"
#include "types.h"

int main(int argc, char **argv)
{
	hipDeviceReset();
	// initialize the computational domain
	domain dom_info;

	// initialize the parameters of the simulation
	parameterDB paramDB;

	// read input .yaml files
	io::readInputs(argc, argv, paramDB, dom_info);

	//print simulation info
	io::printSimulationInfo(paramDB, dom_info);

	// create and initialize the flow solver, I think this can be simplified/streamlined now that there is only one solver
	NavierStokesSolver *solver = 0;
	solverType st = paramDB["simulation"]["SolverType"].get<solverType>();
	switch(st)
	{
	case NAVIERSTOKES:
		solver = new NavierStokesSolver(&paramDB, &dom_info);
		break;
	case FADLUN:
		solver = new fadlunModified(&paramDB, &dom_info);
		break;
	case OSC:
		solver = new luo_iter(&paramDB, &dom_info);
		break;
	case LUO:
		solver = new luoIBM(&paramDB, &dom_info);
		break;
	//case FSI:
		//solver = new FSI(&paramDB, &dom_info);
		//break;
	}
	solver->initialise();

	//prints to output and files
	io::printDeviceMemoryUsage();
	io::writeInfoFile(paramDB, dom_info);

	// time-step loop
	while (!solver->finished())
	{
		solver->stepTime();
		solver->writeData();
	}

	solver->shutDown();
}
