#include "hip/hip_runtime.h"
/***************************************************************************//**
 * \file NavierStokesSolver.cu
 * \author Anush Krishnan (anush@bu.edu), Christopher Minar (minarc@oreonstate.edu)
 * \brief Implementation of the methods of the class \c NavierStokesSolver.
 */

#include "NavierStokesSolver.h"
#include "NavierStokes/kernels/initialise.h"
#include <sys/stat.h>
#include <io/io.h>
#include <cusp/precond/aggregation/smoothed_aggregation.h>//flag
#include <cusp/krylov/cg.h>//flag
#include <cusp/krylov/bicgstab.h>
#include <cusp/krylov/gmres.h>//flag
#include <cusp/krylov/bicg.h>//flag
#include <cusp/print.h>//flag
#include <cusp/blas/blas.h>//flag

#include <iostream>
#include <fstream>

//##############################################################################
//                              INITIALISE
//##############################################################################

/**
 * \brief Initializes parameters, arrays and matrices required for the simulation.
 */

/* initialise the simulation
 *
 */
void NavierStokesSolver::initialise()
{
	logger.startTimer("initialise");
	initialiseNoBody();
	initialiseLHS();
	logger.stopTimer("initialise");
}

/* intialise the parts of the simulation that share no similarities to a body solver
 *
 */
void NavierStokesSolver::initialiseNoBody()
{
	printf("Initalising...\n");
	//////////////////////////////////////////////////////////////////////////////////////////////////
	//Resize and Cast
	//////////////////////////////////////////////////////////////////////////////////////////////////
	NavierStokesSolver::cast();

	std::cout << "NavierStokesSolver: Arrays resized and cast!" << std::endl;
	//////////////////////////////////////////////////////////////////////////////////////////////////
	//COMMON STUFF
	//////////////////////////////////////////////////////////////////////////////////////////////////
	// initial values of timeStep
	timeStep = (*paramDB)["simulation"]["startStep"].get<int>();

	// creates directory
	std::string folder = (*paramDB)["inputs"]["caseFolder"].get<std::string>();
	io::makeDirectory(folder);

	// writes the grids information to a file
	io::writeGrid(folder, *domInfo);

	std::cout << "NavierStokesSolver: Initialised common stuff!" << std::endl;
	////////////////////////////////////////////////////////////////////////////////////////////////
	//ARRAYS
	////////////////////////////////////////////////////////////////////////////////////////////////
	cusp::blas::fill(rhs2, 0);//flag
	cusp::blas::fill(uhat, 0);//flag
	cusp::blas::fill(Nold, 0);//flag
	cusp::blas::fill(N, 0);//flag
	std::cout<<"NavierStokesSolver: Initialised Arrays!" <<std::endl;

	///////////////////////////////////////////////////////////////////////////////////////////////
	//Initialise velocity arrays
	///////////////////////////////////////////////////////////////////////////////////////////////

	double	uInitial = (*paramDB)["flow"]["uInitial"].get<double>(),
			vInitial = (*paramDB)["flow"]["vInitial"].get<double>(),
			uPerturb = (*paramDB)["flow"]["uPerturb"].get<double>(),
			vPerturb = (*paramDB)["flow"]["vPerturb"].get<double>(),
			xmin = domInfo->x[0],
			xmax = domInfo->x[nx-1],
			ymin = domInfo->y[0],
			ymax = domInfo->y[ny-1];
	const int blocksize = 256;
	dim3 dimGridU( int( ((nx-1)*ny-0.5)/blocksize ) +1, 1);
	dim3 dimBlock(blocksize, 1);
	dim3 dimGridV( int( (nx*(ny-1)-0.5)/blocksize ) +1, 1);

	kernels::initialiseU<<<dimGridU,dimBlock>>>(u_r, xu_r, yu_r, uInitial, uPerturb, M_PI, xmax, xmin, ymax, ymin, nx, ny);
	kernels::initialiseV<<<dimGridV,dimBlock>>>(u_r, xv_r, yv_r, vInitial, vPerturb, M_PI, xmax, xmin, ymax, ymin, nx, ny);

	uhat=u;

	std::cout<<"NavierStokesSolver: Initialised Velocities!" <<std::endl;

	/////////////////////////////////////////////////////////////////////////////////////////////////////////////
	//Initialise boundary condition arrays
	/////////////////////////////////////////////////////////////////////////////////////////////////////////////
	boundaryCondition
		**bcInfo
		 = (*paramDB)["flow"]["boundaryConditions"].get<boundaryCondition **>();

	//Top and Bottom
	for(int i=0; i<nx-1; i++)
	{
		bc[YMINUS][i] = bcInfo[YMINUS][0].value;
		bc[YPLUS][i]  = bcInfo[YPLUS][0].value;
		bc[YMINUS][i+nx-1]	= bcInfo[YMINUS][1].value;
		bc[YPLUS][i+nx-1]	= bcInfo[YPLUS][1].value;
	}
	bc[YMINUS][2*nx-2]	= bcInfo[YMINUS][1].value;
	bc[YPLUS][2*nx-2]	= bcInfo[YPLUS][1].value;

	/// Left and Right
	for(int i=0; i<ny-1; i++)
	{
		bc[XMINUS][i] = bcInfo[XMINUS][0].value;
		bc[XPLUS][i]  = bcInfo[XPLUS][0].value;
		bc[XMINUS][i+ny] = bcInfo[XMINUS][1].value;
		bc[XPLUS][i+ny]  = bcInfo[XPLUS][1].value;
	}
	bc[XMINUS][ny-1] = bcInfo[XMINUS][0].value;
	bc[XPLUS][ny-1]  = bcInfo[XPLUS][0].value;

	std::cout << "NavierStokesSolver: Initialised boundary conditions!" << std::endl;

	////////////////////////////////////////////////////////////////////////////////////////////////
	//OUTPUT
	////////////////////////////////////////////////////////////////////////////////////////////////
	cfl_max = 0;
	std::stringstream outiter;
	outiter << folder << "/iterations";
	iterationsFile.open(outiter.str().c_str());
}

/*
 * Initialise the left hand sides of the velocity and poission solvers
 * create preconditioners
 */
void NavierStokesSolver::initialiseLHS()
{
	generateLHS1();
	generateLHS2();
	std::cout << "NavierStokesSolver: Initialised LHS!" << std::endl;

	PC.generate1(LHS1, (*paramDB)["velocitySolve"]["preconditioner"].get<preconditionerType>());
	PC.generate2(LHS2, (*paramDB)["PoissonSolve"]["preconditioner"].get<preconditionerType>());
	std::cout << "NavierStokesSolver: Initialised Preconditioners!" << std::endl;
}

//##############################################################################
//                            TIME STEPPING
//##############################################################################

/**
 * \brief Calculates the variables at the next time step.
 */
void NavierStokesSolver::stepTime()
{
	//1: Solve for intermediate velocity
	generateRHS1();
	solveIntermediateVelocity();

	//2: Solve for pressure correction
	generateRHS2();
	solvePoisson();

	//3: Project velocity
	velocityProjection();

	//4: update time
	timeStep++;
	CFL();
	if (timeStep%(*paramDB)["simulation"]["nt"].get<int>() == 0)
	{
		std::cout<<"Maximun CFL: " << cfl_max << std::endl;
		std::cout<<"Expected CFL: " << (*paramDB)["simulation"]["dt"].get<double>()*bc[XMINUS][0]/domInfo->mid_h << std::endl;
		std::cout<<"CFL I: " << cfl_I << std::endl;
		std::cout<<"CFL J: " << cfl_J << std::endl;
		std::cout<<"CFL ts: " << cfl_ts << std::endl;
	}
	//std::cout<<"Timestep: "<<timeStep<<"\n";
}

/**
 * \brief Evaluates the condition required to stop the simulation.
 *
 * \return a Boolean to continue or stop the simulation
 */
bool NavierStokesSolver::finished()
{
	int nt = (*paramDB)["simulation"]["nt"].get<int>();
	return (timeStep < nt) ? false : true;
}

/**
 * \brief Constructor. Copies the database and information about the computational grid.
 *
 * \param pDB database that contains all the simulation parameters
 * \param dInfo information related to the computational grid
 */
NavierStokesSolver::NavierStokesSolver(parameterDB *pDB, domain *dInfo)
{
	paramDB = pDB;
	domInfo = dInfo;
}

//##############################################################################
//                           LINEAR SOLVES
//##############################################################################

/**
 * \brief Solves for the intermediate velocity.
 */

void NavierStokesSolver::solveIntermediateVelocity()
{
	logger.startTimer("Intermediate Velocity Solve");
	int  maxIters = (*paramDB)["velocitySolve"]["maxIterations"].get<int>();
	double relTol = (*paramDB)["velocitySolve"]["tolerance"].get<double>();

	cusp::monitor<double> sys1Mon(rhs1,maxIters,relTol);//flag currently this takes much more time than it should.
	cusp::krylov::bicgstab(LHS1, uhat, rhs1, sys1Mon, *PC.PC1);

	iterationCount1 = sys1Mon.iteration_count();

	if (!sys1Mon.converged())
	{
		std::cout << "ERROR: Solve for uhat failed at time step " << timeStep << std::endl;
		std::cout << "Iterations   : " << iterationCount1 << std::endl;
		std::cout << "Residual norm: " << sys1Mon.residual_norm() << std::endl;
		std::cout << "Tolerance    : " << sys1Mon.tolerance() << std::endl;
		crash();
		std::exit(-1);
	}

	logger.stopTimer("Intermediate Velocity Solve");
}

/**
 * \brief Solves the Poisson system for the pressure (and the body forces if immersed body).
 */

void NavierStokesSolver::solvePoisson()
{
	logger.startTimer("Poisson Solve");

	int  maxIters = (*paramDB)["PoissonSolve"]["maxIterations"].get<int>();
	double relTol   = (*paramDB)["PoissonSolve"]["tolerance"].get<double>();

	cusp::monitor<double> sys2Mon(rhs2, maxIters, relTol);
	cusp::krylov::bicgstab(LHS2, pressure, rhs2, sys2Mon, *PC.PC2);

	iterationCount2 = sys2Mon.iteration_count();
	if (!sys2Mon.converged())
	{
		std::cout << "ERROR: Solve for pressure failed at time step " << timeStep << std::endl;
		std::cout << "Iterations   : " << iterationCount2 << std::endl;
		std::cout << "Residual norm: " << sys2Mon.residual_norm() << std::endl;
		std::cout << "Tolerance    : " << sys2Mon.tolerance() << std::endl;
		crash();
		std::exit(-1);
	}

	logger.stopTimer("Poisson Solve");
}

//##############################################################################
//                               OUTPUT
//##############################################################################

/**
 * prints an array
 * param value the array
 * param type type of array, p, x, y
 */

void NavierStokesSolver::crash()
{
}

void NavierStokesSolver::arrayprint(cusp::array1d<double, cusp::device_memory> value, std::string name, std::string type, int time)
{
	if (timeStep != time && time > 0) //set time to a negative number to always print
		return;
	logger.startTimer("output");

	int x_length = nx;
	int y_length = ny;
	int i = 0;
	int row_length = nx;
	int pad = 0;
	if (type == "x")
	{
		x_length = nx-1;
		row_length = (nx-1);
		pad = 0;
	}
	if (type == "y")
	{
		y_length = ny-1;
		row_length = nx;
		pad = (nx-1)*ny;
	}

	std::ofstream myfile;
	std::string folder = (*paramDB)["inputs"]["caseFolder"].get<std::string>();
	std::stringstream out;
	std::stringstream convert; convert << "/output/" << timeStep << name << ".csv";
	std::string folder_name = convert.str();
	out<<folder<<folder_name;
	myfile.open(out.str().c_str());
	myfile<<name<<"\n";
	for (int J = 0; J < y_length; J++)
	{
		for (int I = 0; I < x_length; I++)
		{
			i = row_length*J + I + pad;
			myfile<<round(10000*value[i])/10000;
			//myfile<<value[i];
			myfile<<'\t';
		}
		myfile<<"\n";
	}
	myfile.close();
	std::cout<<"printed "<<name <<"\n";
	logger.stopTimer("output");
}

/**
 * \brief Writes numerical solution at current time-step,
 *        as well as the number of iterations performed in each solver.
 */
void NavierStokesSolver::writeCommon()
{

	int nsave = (*paramDB)["simulation"]["nsave"].get<int>();
	std::string folder = (*paramDB)["inputs"]["caseFolder"].get<std::string>();

	// write the velocity fluxes and the pressure values
	if (timeStep % nsave == 0)
		io::writeData(folder, timeStep, uhat, pressure, *domInfo);//, *paramDB);

	// write the number of iterations for each solve
	iterationsFile << timeStep << '\t' << iterationCount1 << '\t' << iterationCount2 << std::endl;
}

/**
 * \brief Writes data into files.
 */
void NavierStokesSolver::writeData()
{
	logger.startTimer("output");

	writeCommon();

	logger.stopTimer("output");
}

/**
 * \brief Prints timing information and closes the different files.
 */
void NavierStokesSolver::shutDown()
{
	io::printTimingInfo(logger);
	iterationsFile.close();
}
