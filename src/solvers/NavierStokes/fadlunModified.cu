/***************************************************************************//**
 * \file  fadlunModified.cu
 * \author Christopher Minar (minarc@oregonstate.edu)
 * \based on code by Anush Krishnan (anush@bu.edu)
 * \brief Declaration of the class oscCylinder.
 */

#include "fadlunModified.h"
#include <sys/stat.h>

/**
 * \brief Constructor. Copies the database and information about the computational grid.
 *
 * \param pDB database that contains all the simulation parameters
 * \param dInfo information related to the computational grid
 */
fadlunModified::fadlunModified(parameterDB *pDB, domain *dInfo)
{
	paramDB = pDB;
	domInfo = dInfo;
}

/*
 * Initialise the solver
 */
void fadlunModified::initialise()
{

	NavierStokesSolver::initialiseNoBody();
	NavierStokesSolver::logger.startTimer("initialise");

	int nx = NavierStokesSolver::domInfo->nx,
		ny = NavierStokesSolver::domInfo->ny;

	int numUV = (nx-1)*ny + nx*(ny-1);
	int numP  = nx*ny;
	////////////////////////////////////////////////////////////////////////////////////////////////
	//ARRAYS
	////////////////////////////////////////////////////////////////////////////////////////////////
	//tagpoints, size uv, device
	tags.resize(numUV);//used in lhs1
	tagsOld.resize(numUV);
	tagsPOld.resize(numP);
	tags2.resize(numUV);//used in lhs1
	tagsIn.resize(numUV);//used in lhs1
	distance_from_intersection_to_node.resize(numUV);
	distance_between_nodes_at_IB.resize(numUV);
	uv.resize(numUV);

	//tagpoints, size np, device
	tagsP.resize(numP);//flag
	tagsPOut.resize(numP);//flag
	distance_from_u_to_body.resize(numP);
	distance_from_v_to_body.resize(numP);
	test.resize(numP); //flag

	cusp::blas::fill(tagsOld,-1);
	cusp::blas::fill(tagsPOld,-1);

	////////////////////////////////////////////////////////////////////////////////////////////////
	//Initialize Bodies
	////////////////////////////////////////////////////////////////////////////////////////////////
	B.initialise((*paramDB), *domInfo);
	std::cout << "Initialised bodies!" << std::endl;

	/////////////////////////////////////////////////////////////////////////////////////////////////
	//TAG POINTS
	/////////////////////////////////////////////////////////////////////////////////////////////////
	tagPoints();
	std::cout << "Tagged points!" << std::endl;

	/////////////////////////////////////////////////////////////////////////////////////////////////
	//LHS
	/////////////////////////////////////////////////////////////////////////////////////////////////
	initialiseLHS();

	/////////////////////////////////////////////////////////////////////////////////////////////////
	//OUTPUT
	/////////////////////////////////////////////////////////////////////////////////////////////////
	parameterDB  &db = *NavierStokesSolver::paramDB;
	std::string folder = db["inputs"]["caseFolder"].get<std::string>();
	std::stringstream out;
	out << folder << "/forces";
	forceFile.open(out.str().c_str());

	logger.stopTimer("initialise");
}

/*
 * Initliase the LHS matrices
 */
void fadlunModified::initialiseLHS()
{
	parameterDB  &db = *NavierStokesSolver::paramDB;
	int nx = domInfo->nx,
		ny = domInfo->ny,
		numUV = (nx-1)*ny + (ny-1)*nx;
	LHS1.resize(numUV, numUV, (nx-1)*ny*5 - 2*ny-2*(nx-1)       +        (ny-1)*nx*5 - 2*(ny-1) - 2*nx);
	LHS2.resize(nx*ny, nx*ny, 5*nx*ny - 2*ny-2*nx); //flag this should have some zero terms in it because no nodes are being removing to account for the different stencil at the body
	generateLHS1();
	generateLHS2();

	NavierStokesSolver::PC.generate(NavierStokesSolver::LHS1,NavierStokesSolver::LHS2, db["velocitySolve"]["preconditioner"].get<preconditionerType>(), db["PoissonSolve"]["preconditioner"].get<preconditionerType>());
	std::cout << "Assembled FADLUN LHS matrices!" << std::endl;
}


/**
 * \brief Writes data into files.
 */
void fadlunModified::writeData()
{

	parameterDB  &db = *NavierStokesSolver::paramDB;
	double dt  = db["simulation"]["dt"].get<double>();

	logger.startTimer("output");

	writeCommon();
	calculateForce();
	if (NavierStokesSolver::timeStep == 0)
		forceFile<<"timestep\tFx\tFxX\tFxY\tFxU\tFy\n";
	forceFile << timeStep*dt << '\t' << B.forceX[0] << '\t'<<fxx<<"\t"<<fxy<<"\t"<<fxu<<"\t" << B.forceY[0] << std::endl;

	logger.stopTimer("output");
}

/**
 * \brief Writes numerical solution at current time-step,
 *        as well as the number of iterations performed in each solver.
 */
void fadlunModified::writeCommon()
{
	NavierStokesSolver::writeCommon();
	parameterDB  &db = *NavierStokesSolver::paramDB;
	int nsave = db["simulation"]["nsave"].get<int>();
	std::string folder = db["inputs"]["caseFolder"].get<std::string>();

	// write body output
	if (timeStep % nsave == 0)
	{
		B.writeToFile(folder, NavierStokesSolver::timeStep);
	}
}

void fadlunModified::stepTime()
{
	generateRHS1();
	solveIntermediateVelocity();

	generateRHS2();
	solvePoisson();

	velocityProjection();

	//std::cout<<timeStep<<std::endl;
	timeStep++;
}

/**
 * \brief Prints timing information and closes the different files.
 */
void fadlunModified::shutDown()
{
	NavierStokesSolver::shutDown();
	forceFile.close();
}

#include "FadlunModified/intermediateVelocity.inl"
#include "FadlunModified/intermediatePressure.inl"
#include "FadlunModified/projectVelocity.inl"
#include "FadlunModified/tagpoints.inl"
#include "FadlunModified/calculateForce.inl"
#include "FadlunModified/checkTags.inl"
